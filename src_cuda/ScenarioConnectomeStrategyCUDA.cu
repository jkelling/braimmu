#include "hip/hip_runtime.h"
#include "ScenarioConnectomeStrategyCUDA.h"

#include "scenario_connectome.h"

#include "hipError_t.h"

#include <hip/hip_runtime.h>

#include <algorithm>

using namespace std;

struct Coord {
  int x;
  int y;
  int z;
};

__constant__ ScenarioConnectome::properties prop;
__constant__ int nvl[ndim];
__constant__ ScenarioConnectomeStrategyCUDA::AllocPitch pitch;

static __device__ constexpr int tissue(int type)
{
	return 1<<type;
}

static __global__ void derivativeKernel(const double* agent, double* agent2, const int* type, const ScenarioConnectomeStrategyCUDA::array_properties arr_prop, int nall,
double dt, int step);

static __global__ void updateKernel(double* agent, const double* deriv, const int* type, const ScenarioConnectomeStrategyCUDA::array_properties arr_prop, double dt, int nall);

size_t ScenarioConnectomeStrategyCUDA::nvx()
{
	return m_this->nvl[0]+2;
}
size_t ScenarioConnectomeStrategyCUDA::nvyz()
{
	return (m_this->nvl[1]+2)*(m_this->nvl[2]+2);
}

ScenarioConnectomeStrategyCUDA::ScenarioConnectomeStrategyCUDA(ScenarioConnectome* pthis)
	: ScenarioConnectomeAbstractStrategy(pthis)
{
	const size_t width = this->nvx();
	const size_t height = this->nvyz();
	CUDA_SAFE_CALL(
		hipMallocPitch(&arr_prop.Dtau, &m_allocPitch.pDouble, width*sizeof(double), height*ndim)
		);
	std::cout << "pitch Dtau: " << m_allocPitch.pDouble << std::endl;
	CUDA_SAFE_CALL(
		hipMallocPitch(&agent, &m_allocPitch.pDouble, width*sizeof(double), height*ScenarioConnectomeAgents::num_agents)
		);
	std::cout << "pitch agent: " << m_allocPitch.pDouble << std::endl;
	CUDA_SAFE_CALL(
		hipMallocPitch(&deriv, &m_allocPitch.pDouble, width*sizeof(double), height*ScenarioConnectomeAgents::num_agents)
		);
	std::cout << "pitch deriv: " << m_allocPitch.pDouble << std::endl;
	CUDA_SAFE_CALL(
		hipMallocPitch(&type, &m_allocPitch.pInt, width*sizeof(int), height)
		);
	std::cout << "pitch type: " << m_allocPitch.pInt << std::endl;

	m_allocPitch.pDouble /= sizeof(double);
	m_allocPitch.pInt /= sizeof(int);

	CUDA_SAFE_CALL(
		hipMemcpyToSymbol(HIP_SYMBOL(prop), &m_this->prop, sizeof(ScenarioConnectome::properties))
		);
	CUDA_SAFE_CALL(
		hipMemcpyToSymbol(HIP_SYMBOL(nvl), (void*)m_this->nvl.data(), sizeof(int)*m_this->nvl.size())
		);
	CUDA_SAFE_CALL(
		hipMemcpyToSymbol(HIP_SYMBOL(pitch), &m_allocPitch, sizeof(AllocPitch))
		);
}

ScenarioConnectomeStrategyCUDA::~ScenarioConnectomeStrategyCUDA()
{
	CUDA_SAFE_CALL(
		hipFree(arr_prop.Dtau)
		);
	CUDA_SAFE_CALL(
		hipFree(agent)
		);
	CUDA_SAFE_CALL(
		hipFree(deriv)
		);
	CUDA_SAFE_CALL(
		hipFree(type)
		);
}

void ScenarioConnectomeStrategyCUDA::push()
{
	const size_t width = this->nvx();
	const size_t height = this->nvyz();
	for(int a = 0; a < ndim; ++a)
	{
		CUDA_SAFE_CALL(
			/*hipMemcpy(arr_prop.Dtau+m_this->nall*a, m_this->arr_prop.Dtau[a].data(), m_this->nall*sizeof(double), hipMemcpyHostToDevice)*/
			hipMemcpy2D(arr_prop.Dtau+height*m_allocPitch.pDouble*a, m_allocPitch.pDouble*sizeof(double)
				, m_this->arr_prop.Dtau[a].data(), width*sizeof(double), width*sizeof(double), height
				, hipMemcpyHostToDevice)
			);
	}
	for(int a = 0; a < ScenarioConnectomeAgents::num_agents; ++a)
	{
		CUDA_SAFE_CALL(
			/*hipMemcpy(agent+m_this->nall*a, m_this->agent[a].data(), m_this->nall*sizeof(double), hipMemcpyHostToDevice)*/
			hipMemcpy2D(agent+height*m_allocPitch.pDouble*a, m_allocPitch.pDouble*sizeof(double)
				, m_this->agent[a].data(), width*sizeof(double), width*sizeof(double), height
				, hipMemcpyHostToDevice)
			);
	}
	CUDA_SAFE_CALL(
		hipMemsetAsync(type, 0, sizeof(int)*m_allocPitch.pInt*height)
		);
	CUDA_SAFE_CALL(
		/*hipMemcpy(type, m_this->type.data(), m_this->nall*sizeof(int), hipMemcpyHostToDevice)*/
		hipMemcpy2D(type, m_allocPitch.pInt*sizeof(int)
			, m_this->type.data(), width*sizeof(int), width*sizeof(int), height
			, hipMemcpyHostToDevice)
		);
}

void ScenarioConnectomeStrategyCUDA::pop()
{
	const size_t width = this->nvx();
	const size_t height = this->nvyz();
	for(int a = 0; a < ScenarioConnectomeAgents::num_agents; ++a)
	{
		CUDA_SAFE_CALL(
			/*hipMemcpy(m_this->agent[a].data(), agent + m_this->nall*a, m_this->nall*sizeof(double), hipMemcpyDeviceToHost)*/
			hipMemcpy2D(m_this->agent[a].data(), width*sizeof(double)
				, agent+height*m_allocPitch.pDouble*a, m_allocPitch.pDouble*sizeof(double)
				, width*sizeof(double), height
				, hipMemcpyDeviceToHost)
			);
		CUDA_SAFE_CALL(
			/*hipMemcpy(m_this->deriv[a].data(), deriv+m_this->nall*a, m_this->nall*sizeof(double), hipMemcpyDeviceToHost)*/
			hipMemcpy2D(m_this->deriv[a].data(), width*sizeof(double)
				, deriv+height*m_allocPitch.pDouble*a, m_allocPitch.pDouble*sizeof(double)
				, width*sizeof(double), height
				, hipMemcpyDeviceToHost)
			);
	}
}

using namespace ScenarioConnectomeAgents;

/*static __global__ void zeroKernel(double* data, size_t n)
{
	int* tmp = (int*)data;
	const int id = threadIdx.x + blockIdx.x*blockDim.x;
	tmp[id] = 0;
	tmp[id+n] = 0;
}*/

/* ----------------------------------------------------------------------*/
void ScenarioConnectomeStrategyCUDA::derivatives() {

	static constexpr int BLOCK_DIM = 128;
  // set derivatives of all voxels to zero
#if 0
	{
		const size_t n = ScenarioConnectomeAgents::num_agents*m_this->nall;
		zeroKernel<<<n/BLOCK_DIM + (n%BLOCK_DIM>0), BLOCK_DIM>>>(deriv, n);
	}
#endif

  const dim3 blocks(m_this->nvl[0]/BLOCK_DIM + (m_this->nvl[0]%BLOCK_DIM>0), m_this->nvl[1], m_this->nvl[2]);
  derivativeKernel<<<blocks, BLOCK_DIM>>>(agent, deriv, type, arr_prop, m_this->nall,m_this->dt, m_this->step);
  // derivativeKernel<<<m_this->nall/BLOCK_DIM + (m_this->nall%BLOCK_DIM>0), BLOCK_DIM>>>(
  //    agent, deriv, type, arr_prop, m_this->nall,m_this->dt, m_this->step);
  std::swap(agent, deriv);
}

static __device__ int find_id(int i, int j, int k)
{
	return i + (pitch.pDouble) * (j + (nvl[1] + 2) * k);
}

static __device__ Coord find_coord(int i)
{
  Coord coord;
  coord.x = i % (nvl[0] + 2);
  i /= (nvl[0] + 2);
  coord.y = i % (nvl[1] + 2);
  coord.z = i / (nvl[1] + 2);
  
  return coord;

}

static __global__ void derivativeKernel(const double* agent, double* agent2,
    const int* type, const ScenarioConnectomeStrategyCUDA::array_properties arr_prop,
    int nall, double dt, int step)
{
  const int ii = threadIdx.x + blockDim.x*blockIdx.x;
  const int jj = blockIdx.y +1;
  const int kk = blockIdx.z +1;
  nall = pitch.pDouble * (nvl[1]+2)*(nvl[2]+2);
  if(ii > 0 && ii < nvl[0]+1)
  {
    const int i = find_id(ii,jj,kk);

    const int t = type[i];
    if (t & tissue(EMP)) return;

  // direct function or time derivatives

	const auto ag_sAb = agent[sAb * nall + i];
	const auto ag_fAb = agent[fAb * nall + i];
	const auto ag_phr = agent[phr * nall + i];

  double dum = prop.kp * ag_sAb * ag_fAb
				 + prop.kn * ag_sAb * ag_sAb;

  agent2[sAb * nall + i] = ag_sAb + dt* ((t & tissue(CSF))
    // sAb, fAb, and tau efflux from CSF
    ? -prop.es * ag_sAb
    // in parenchyma (WM and GM)
     : agent[neu * nall + i] * agent[cir * nall + i]
      - dum
      - prop.ds * agent[mic * nall + i] * ag_sAb);

  agent2[fAb * nall + i] = ag_fAb + dt* ((t & tissue(CSF))
    ? -prop.es * ag_fAb
    : dum
      - prop.df * agent[mic * nall + i] * ag_fAb);

  dum = prop.ktau * ag_phr;
  // tau protein phosphorylation due to fAb and neu
  agent2[phr * nall + i] = ag_phr + dt* ((t & tissue(CSF))
    ? -prop.ephi * ag_phr
    : prop.kphi * ag_fAb * agent[neu * nall + i]
      - dum);

  // tau tangle formation from phosphorylated tau
  agent2[tau * nall + i] = agent[tau * nall + i] + dt* ((t & tissue(CSF)) ? 0 : dum) ;

  // neuronal death due to tau aggregation
  agent2[neu * nall + i] = agent[neu * nall + i] + dt* ((t & tissue(CSF)) ? 0 : -prop.dnt * agent[tau * nall + i] * agent[neu * nall + i]);

  // astrogliosis
  dum = ag_fAb * agent[mic * nall + i];
  agent2[ast * nall + i] = agent[ast * nall + i] + dt* ((t & tissue(CSF)) ? 0 : prop.ka * (dum / (dum + prop.Ha) - agent[ast * nall + i]));

  // circadian rhythm
  // !!inverse CSF check!!
  agent2[cir * nall + i] = agent[cir * nall + i] + dt* ((!(t & tissue(CSF)) && (prop.c_cir > 0))
    ? - prop.C_cir * prop.c_cir * prop.omega_cir
      * sin(prop.omega_cir * dt * step)
    : 0);

      double de_mic = 0.;
      const double ag_mic = agent[mic * nall + i];
      #pragma unroll
		  for(int s = -1; s <= 1; s+=2)
			  for (int d=0; d < 3; d+=1) {
			    const int j = find_id(ii +s*(d==0),jj +s*(d==1),kk +s*(d==2));

			    if (type[j] & tissue(EMP)) continue;

			    const double del_phr = ag_phr - agent[phr * nall + j];

			    // diffusion of tau
			    agent2[phr * nall + i] -= 0.5 * (arr_prop.Dtau[ nall * d + i] + arr_prop.Dtau[nall * d + j]) * del_phr *dt;

			    const double del_sAb = ag_sAb - agent[sAb * nall + j];

			    // diffusion of sAb
			    agent2[sAb * nall + i] -= prop.D_sAb * del_sAb *dt;

			    // only in parenchyma
			    if (t & tissue(WM) || t & tissue(GM))
				  if (type[j] & tissue(WM) || type[j] & tissue(GM)) {
				    const double del_fAb = ag_fAb - agent[fAb * nall + j];
				    const double del_mic = ag_mic - agent[mic * nall + j];

				    // migration of microglia toward higher sAb concentrations
				    de_mic += prop.cs * del_sAb * ((del_sAb > 0.0) ? agent[mic * nall + j] : ag_mic);

				    // migration of microglia toward higher fAb concentrations
            de_mic += prop.cf * del_fAb * ((del_fAb > 0.0) ? agent[mic * nall + j] : ag_mic);

				    // diffusion of microglia
				    de_mic -= prop.D_mic * del_mic;
		      }
		    }
		agent2[mic * nall + i] = ag_mic + de_mic *dt;
	  }
}

#include "ScenarioConnectomeStrategyCUDANewton.cu"
