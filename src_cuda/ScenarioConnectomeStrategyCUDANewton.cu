#include "hip/hip_runtime.h"
#include "ScenarioConnectomeStrategyCUDA.h"
#include "ScenarioConnectomeStrategyCUDANewton.h"

#include "scenario_connectome.h"

#include "hipError_t.h"

#include <hip/hip_runtime.h>

static __global__ void derivativeKernelNewton(const double* agent, double* deriv, const int* type,
                                        const ScenarioConnectomeStrategyCUDA::array_properties arr_prop,
                                        int nall, double dt, int step, int parity);

/* ----------------------------------------------------------------------*/
void ScenarioConnectomeStrategyCUDANewton::derivatives() {

  static constexpr int BLOCK_DIM = 128;
  // set derivatives of all voxels to zero
  CUDA_SAFE_CALL( hipMemsetAsync(deriv, 0, ScenarioConnectomeAgents::num_agents*sizeof(double)*m_this->nall) );

  const dim3 blocks( (m_this->nvl[0] + 1)/2/BLOCK_DIM + (( (m_this->nvl[0] + 1)/2)%BLOCK_DIM>0), m_this->nvl[1], m_this->nvl[2]);
  derivativeKernelNewton<<<blocks, BLOCK_DIM>>>(agent, deriv, type, arr_prop, m_this->nall,m_this->dt, m_this->step, 0);
  derivativeKernelNewton<<<blocks, BLOCK_DIM>>>(agent, deriv, type, arr_prop, m_this->nall,m_this->dt, m_this->step, 1);
  //derivativeKernelNewton<<<m_this->nall/BLOCK_DIM + (m_this->nall%BLOCK_DIM>0), BLOCK_DIM>>>(agent, deriv, type, arr_prop, m_this->nall,m_this->dt, m_this->step, 0);
  //derivativeKernelNewton<<<m_this->nall/BLOCK_DIM + (m_this->nall%BLOCK_DIM>0), BLOCK_DIM>>>(agent, deriv, type, arr_prop, m_this->nall,m_this->dt, m_this->step, 1);
}

static __device__ int findParity(Coord coord)
{
  return (int) ( (coord.z ^ coord.y ^ (coord.x % 2) ) & 1 );
}

static __global__ void derivativeKernelNewton(const double* agent, double* deriv, const int* type,
                                        const ScenarioConnectomeStrategyCUDA::array_properties arr_prop,
                                        int nall, double dt, int step, int parity)
{
  Coord coord;

  coord.x = threadIdx.x + blockDim.x*blockIdx.x;
  coord.y = blockIdx.y +1;
  coord.z = blockIdx.z +1;

  // choose the right parity
  coord.x = (( (coord.z ^ coord.y) ^ parity) & 1) + 2 * coord.x;
  
  if(coord.x < nvl[0]+1 && coord.x > 0) {
    const int i = find_id(coord.x,coord.y,coord.z);

    //printf("id=%li , coords = %i %i %i parity = %i \n", i, coord.x, coord.y, coord.z, findParity(coord));

    if (type[i] & tissue(EMP)) return;

    // sAb, fAb, and tau efflux from CSF
    if (type[i] & tissue(CSF)) {
      deriv[sAb * nall + i] -= prop.es * agent[sAb * nall + i];
      deriv[fAb * nall + i] -= prop.es * agent[fAb * nall + i];
      deriv[phr * nall + i] -= prop.ephi * agent[phr * nall + i];
    }

    // in parenchyma (WM and GM)
    else {
      double dum = prop.kp * agent[sAb * nall + i] * agent[fAb * nall + i]
                     + prop.kn * agent[sAb * nall + i] * agent[sAb * nall + i];

      // sAb
      deriv[sAb * nall + i] += agent[neu * nall + i] * agent[cir * nall + i]
                            - dum
                            - prop.ds * agent[mic * nall + i] * agent[sAb * nall + i];
      // fAb
      deriv[fAb * nall + i] += dum
                            - prop.df * agent[mic * nall + i] * agent[fAb * nall + i];

      dum = prop.ktau * agent[phr * nall + i];

      // tau protein phosphorylation due to fAb and neu
      deriv[phr * nall + i] += prop.kphi * agent[fAb * nall + i] * agent[neu * nall + i]
                            - dum;

      // tau tangle formation from phosphorylated tau
      deriv[tau * nall + i] += dum;

      // neuronal death due to tau aggregation
      deriv[neu * nall + i] -= prop.dnt * agent[tau * nall + i] * agent[neu * nall + i];

      // astrogliosis
      dum = agent[fAb * nall + i] * agent[mic * nall + i];
      deriv[ast * nall + i] = prop.ka * (dum / (dum + prop.Ha) - agent[ast * nall + i]);

      // circadian rhythm
      if (prop.c_cir > 0)
        deriv[cir * nall + i] = - prop.C_cir * prop.c_cir * prop.omega_cir
                            * sin(prop.omega_cir * dt * step);
    }

    for (int d=0; d < 3; d+=1) {
		  const int j = find_id(coord.x + (d==0), coord.y + (d==1), coord.z + (d==2));

		  if (type[j] & tissue(EMP)) continue;

		  double del_phr = agent[phr * nall + i] - agent[phr * nall + j];

		  // diffusion of tau
      double dum = 0.5 * (arr_prop.Dtau[ nall * d + i] + arr_prop.Dtau[nall * d + j]) * del_phr;
      deriv[phr * nall + i] -= dum;
      deriv[phr * nall + j] += dum;

		  double del_sAb = agent[sAb * nall + i] - agent[sAb * nall + j];

		  // diffusion of sAb
		  dum = prop.D_sAb * del_sAb;
      deriv[sAb * nall + i] -= dum;
      deriv[sAb * nall + j] += dum;

      //if (j == 30)
      //printf("i=%li , j= %li, sAb = %g \n", i, j, deriv[sAb * nall + j]);

		  // only in parenchyma
		  if (type[i] & tissue(WM) || type[i] & tissue(GM))
			if (type[j] & tissue(WM) || type[j] & tissue(GM)) {
			  double del_fAb = agent[fAb * nall + i] - agent[fAb * nall + j];
			  double del_mic = agent[mic * nall + i] - agent[mic * nall + j];

			  // migration of microglia toward higher sAb concentrations
			  dum = prop.cs * del_sAb * agent[mic * nall + ((del_sAb > 0.0) ? j : i)];
        deriv[mic * nall + i] += dum;
        deriv[mic * nall + j] -= dum;

			  // migration of microglia toward higher fAb concentrations
        dum = prop.cf * del_fAb * agent[mic * nall + ((del_fAb > 0.0) ? j : i)];
        deriv[mic * nall + i] += dum;
        deriv[mic * nall + j] -= dum;

			  // diffusion of microglia
			  dum = prop.D_mic * del_mic;
        deriv[mic * nall + i] -= dum;
        deriv[mic * nall + j] += dum;
		  }
    }
  }
}
