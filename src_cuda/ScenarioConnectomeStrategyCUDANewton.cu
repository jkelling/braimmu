#include "hip/hip_runtime.h"
#include "ScenarioConnectomeStrategyCUDANewton.h"
#include "scenario_connectome.h"
#include "hipError_t.h"
#include <hip/hip_runtime.h>

using namespace std;

struct Coord {
  int x;
  int y;
  int z;
};

__constant__ ScenarioConnectome::properties prop;
__constant__ int nvl[ndim];

static __device__ constexpr int tissue(int type)
{
	return 1<<type;
}

static __global__ void derivativeKernel(const double* agent, double* deriv, const int* type,
                                        const ScenarioConnectomeStrategyCUDANewton::array_properties arr_prop,
                                        int nall, double dt, int step, int parity);

static __global__ void updateKernel(double* agent, const double* deriv, const int* type,
                                    const ScenarioConnectomeStrategyCUDANewton::array_properties arr_prop,
                                    double dt, int nall);

ScenarioConnectomeStrategyCUDANewton::ScenarioConnectomeStrategyCUDANewton(ScenarioConnectome* pthis)
	: ScenarioConnectomeAbstractStrategy(pthis)
{
	CUDA_SAFE_CALL( hipMalloc(&arr_prop.Dtau, ndim*sizeof(double)*m_this->nall) );
	CUDA_SAFE_CALL(	hipMalloc(&agent, ScenarioConnectomeAgents::num_agents*sizeof(double)*m_this->nall) );
	CUDA_SAFE_CALL( hipMalloc(&deriv, ScenarioConnectomeAgents::num_agents*sizeof(double)*m_this->nall) );
	CUDA_SAFE_CALL(	hipMalloc(&type, sizeof(int)*m_this->nall) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(prop), &m_this->prop, sizeof(ScenarioConnectome::properties)) );
	CUDA_SAFE_CALL(	hipMemcpyToSymbol(HIP_SYMBOL(nvl), (void*)m_this->nvl.data(), sizeof(int)*m_this->nvl.size()) );
}

ScenarioConnectomeStrategyCUDANewton::~ScenarioConnectomeStrategyCUDANewton()
{
	CUDA_SAFE_CALL(	hipFree(arr_prop.Dtau) );
	CUDA_SAFE_CALL(	hipFree(agent) );
	CUDA_SAFE_CALL(	hipFree(deriv) );
	CUDA_SAFE_CALL(	hipFree(type) );
}

void ScenarioConnectomeStrategyCUDANewton::push()
{
	for(int a = 0; a < ndim; ++a)
		CUDA_SAFE_CALL(	hipMemcpy(arr_prop.Dtau+m_this->nall*a, m_this->arr_prop.Dtau[a].data(), m_this->nall*sizeof(double), hipMemcpyHostToDevice) );

	for(int a = 0; a < ScenarioConnectomeAgents::num_agents; ++a)
		CUDA_SAFE_CALL( hipMemcpy(agent+m_this->nall*a, m_this->agent[a].data(), m_this->nall*sizeof(double), hipMemcpyHostToDevice) );

	CUDA_SAFE_CALL( hipMemcpy(type, m_this->type.data(), m_this->nall*sizeof(int), hipMemcpyHostToDevice) );
}

void ScenarioConnectomeStrategyCUDANewton::pop()
{
	for(int a = 0; a < ScenarioConnectomeAgents::num_agents; ++a)
	{
		CUDA_SAFE_CALL( hipMemcpy(m_this->agent[a].data(), agent + m_this->nall*a, m_this->nall*sizeof(double), hipMemcpyDeviceToHost) );
		CUDA_SAFE_CALL( hipMemcpy(m_this->deriv[a].data(), deriv+m_this->nall*a, m_this->nall*sizeof(double), hipMemcpyDeviceToHost));
	}
}

using namespace ScenarioConnectomeAgents;

/* ----------------------------------------------------------------------*/
void ScenarioConnectomeStrategyCUDANewton::derivatives() {

	static constexpr int BLOCK_DIM = 128;
  // set derivatives of all voxels to zero
	CUDA_SAFE_CALL( hipMemsetAsync(deriv, 0, ScenarioConnectomeAgents::num_agents*sizeof(double)*m_this->nall) );

	//const dim3 blocks(m_this->nvl[0]/BLOCK_DIM + (m_this->nvl[0]%BLOCK_DIM>0), m_this->nvl[1], m_this->nvl[2]);
	//derivativeKernel<<<blocks, BLOCK_DIM>>>(agent, deriv, type, arr_prop, m_this->nall,m_this->dt, m_this->step);
  derivativeKernel<<<m_this->nall/BLOCK_DIM + (m_this->nall%BLOCK_DIM>0), BLOCK_DIM>>>(agent, deriv, type, arr_prop, m_this->nall,m_this->dt, m_this->step, 0);
  derivativeKernel<<<m_this->nall/BLOCK_DIM + (m_this->nall%BLOCK_DIM>0), BLOCK_DIM>>>(agent, deriv, type, arr_prop, m_this->nall,m_this->dt, m_this->step, 1);
}

static __device__ int find_id(int i, int j, int k)
{
	return i + (nvl[0] + 2) * (j + (nvl[1] + 2) * k);
}

static __device__ Coord find_coord(int i)
{
  Coord coord;
  coord.x = i % (nvl[0] + 2);
  int dumi = (int) ((i - coord.x) / (nvl[0] + 2));
  coord.y = dumi % (nvl[1] + 2);
  coord.z = (int) ((dumi - coord.y) / (nvl[1] + 2));

  return coord;

}

static __device__ int findParity(Coord coord)
{
  return (int) ( ((coord.z ^ coord.y) ^ (coord.x % 2) ) & 1 );
}

/*

for (int p=0; p<2; p++)
  for (int kk=1; kk<nvl[2]+1; kk++)
    for (int jj=1; jj<nvl[1]+1; jj++)
      for (int ii = (kk^jj^p)&1; ii<nvl[0]+1; ii+=2)

*/

static __global__ void derivativeKernel(const double* agent, double* deriv, const int* type,
                                        const ScenarioConnectomeStrategyCUDANewton::array_properties arr_prop,
                                        int nall, double dt, int step, int parity)
{
  const int i = threadIdx.x + blockDim.x*blockIdx.x;
 
  if(i < nall) {

    Coord coord = find_coord(i);

    if (type[i] & tissue(EMP)) return;

    if (!(parity & findParity(coord))) return;

  //const int ii = threadIdx.x + blockDim.x*blockIdx.x +1;
  //const int jj = blockIdx.y +1;
  //const int kk = blockIdx.z +1;
	//if(ii < nvl[0]+1)
	//{
  //const int i = find_id(ii,jj,kk);
  //if (type[i] & tissue(EMP)) return;
  // direct function or time derivatives

    // sAb, fAb, and tau efflux from CSF
    if (type[i] & tissue(CSF)) {
      deriv[sAb * nall + i] -= prop.es * agent[sAb * nall + i];
      deriv[fAb * nall + i] -= prop.es * agent[fAb * nall + i];
      deriv[phr * nall + i] -= prop.ephi * agent[phr * nall + i];
    }

    // in parenchyma (WM and GM)
    else {
      double dum = prop.kp * agent[sAb * nall + i] * agent[fAb * nall + i]
                     + prop.kn * agent[sAb * nall + i] * agent[sAb * nall + i];

      // sAb
      deriv[sAb * nall + i] += agent[neu * nall + i] * agent[cir * nall + i]
                            - dum
                            - prop.ds * agent[mic * nall + i] * agent[sAb * nall + i];
      // fAb
      deriv[fAb * nall + i] += dum
                            - prop.df * agent[mic * nall + i] * agent[fAb * nall + i];

      dum = prop.ktau * agent[phr * nall + i];

      // tau protein phosphorylation due to fAb and neu
      deriv[phr * nall + i] += prop.kphi * agent[fAb * nall + i] * agent[neu * nall + i]
                            - dum;

      // tau tangle formation from phosphorylated tau
      deriv[tau * nall + i] += dum;

      // neuronal death due to tau aggregation
      deriv[neu * nall + i] -= prop.dnt * agent[tau * nall + i] * agent[neu * nall + i];

      // astrogliosis
      dum = agent[fAb * nall + i] * agent[mic * nall + i];
      deriv[ast * nall + i] = prop.ka * (dum / (dum + prop.Ha) - agent[ast * nall + i]);

      // circadian rhythm
      if (prop.c_cir > 0)
        deriv[cir * nall + i] = - prop.C_cir * prop.c_cir * prop.omega_cir
                            * sin(prop.omega_cir * dt * step);
      }

      for (int d=0; d < 3; d+=1) {
			  const int j = find_id(coord.x + (d==0),coord.y + (d==1), coord.z + (d==2));

			  if (type[j] & tissue(EMP)) continue;

			  double del_phr = agent[phr * nall + i] - agent[phr * nall + j];

			  // diffusion of tau
        double dum = 0.5 * (arr_prop.Dtau[ nall * d + i] + arr_prop.Dtau[nall * d + j]) * del_phr;
        deriv[phr * nall + i] -= dum;
        deriv[phr * nall + j] += dum;

			  double del_sAb = agent[sAb * nall + i] - agent[sAb * nall + j];

			  // diffusion of sAb
        dum = prop.D_sAb * del_sAb;
        deriv[sAb * nall + i] -= dum;
        deriv[sAb * nall + j] += dum;

			  // only in parenchyma
			  if (type[i] & tissue(WM) || type[i] & tissue(GM))
				if (type[j] & tissue(WM) || type[j] & tissue(GM)) {
				  double del_fAb = agent[fAb * nall + i] - agent[fAb * nall + j];
				  double del_mic = agent[mic * nall + i] - agent[mic * nall + j];

				  // migration of microglia toward higher sAb concentrations
				  dum = prop.cs * del_sAb * agent[mic * nall + ((del_sAb > 0.0) ? j : i)];
          deriv[mic * nall + i] += dum;
          deriv[mic * nall + j] -= dum;

				  // migration of microglia toward higher fAb concentrations
          dum = prop.cf * del_fAb * agent[mic * nall + ((del_fAb > 0.0) ? j : i)];
          deriv[mic * nall + i] += dum;
          deriv[mic * nall + j] -= dum;

				  // diffusion of microglia
				  dum = prop.D_mic * del_mic;
          deriv[mic * nall + i] -= dum;
          deriv[mic * nall + j] += dum;

		    }
		  }
	  }
}

/* ----------------------------------------------------------------------*/
void ScenarioConnectomeStrategyCUDANewton::update() {

  using namespace ScenarioConnectomeAgents;

	static constexpr int BLOCK_DIM = 128;
	//const dim3 blocks(m_this->nvl[0]/BLOCK_DIM + (m_this->nvl[0]%BLOCK_DIM>0), m_this->nvl[1], m_this->nvl[2]);
	updateKernel<<<m_this->nall/BLOCK_DIM + (m_this->nall%BLOCK_DIM>0), BLOCK_DIM>>>(agent, deriv, type, arr_prop,m_this->dt, m_this->nall);

}

static __global__ void updateKernel(double* agent, const double* deriv, const int* type, const ScenarioConnectomeStrategyCUDANewton::array_properties arr_prop, double dt, int nall)
{
	const int i = threadIdx.x + blockDim.x*blockIdx.x;
	//const int jj = blockIdx.y +1;
	//const int kk = blockIdx.z +1;
  if(i < nall) {
    
    if (type[i] & tissue(EMP)) return;
    
    // time integration (Euler's scheme)
    for (int ag_id=0; ag_id<ScenarioConnectomeAgents::num_agents; ag_id++)
      agent[ag_id * nall + i] += deriv[ag_id * nall + i] * dt;
  }
}
